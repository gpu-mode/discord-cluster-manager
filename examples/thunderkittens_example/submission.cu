#include "hip/hip_runtime.h"
// Adapted from
// https://github.com/HazyResearch/ThunderKittens/blob/tk_gen/simple_kernels/micro_add/micro.cu
// Test whether TK works on Modal runners.

#include "task.h"
#include "utils.h"
#include <array>
#include <vector>

#include "kittens.cuh"
using namespace kittens;

#define CudaCheckError() __cudaCheckError(__FILE__, __LINE__)
inline void __cudaCheckError(const char *file, const int line) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "cudaCheckError() failed at %s:%i : %s\n", file, line,
            hipGetErrorString(err));
    exit(-1);
  }
  err = hipDeviceSynchronize();
  if (hipSuccess != err) {
    fprintf(stderr, "cudaCheckError() with sync failed at %s:%i : %s\n", file,
            line, hipGetErrorString(err));
    exit(-1);
  }
}

#define NUM_THREADS (kittens::WARP_THREADS) // use 1 warp

#define _row 16
#define _col 32

struct micro_globals {
  using _gl = gl<float, -1, -1, -1, -1, st_fl<_row, _col>>;
  _gl x, o;
};

__global__
__launch_bounds__(NUM_THREADS,
                  1) void micro_tk(const __grid_constant__ micro_globals g) {

  // shared memory
  extern __shared__ alignment_dummy __shm[];
  shared_allocator al((int *)&__shm[0]);
  st_fl<_row, _col>(&x_s) = al.allocate<st_fl<_row, _col>>();
  st_fl<_row, _col>(&o_s) = al.allocate<st_fl<_row, _col>>();

  // register memory
  rt_fl<_row, _col> x_reg_fl;

  // load from HBM to shared
  load(x_s, g.x, {0, 0, 0, 0});
  __syncthreads();

  // load from shared to register
  load(x_reg_fl, x_s);
  __syncthreads();

  // x (dst) = x (src b) + x (src a)
  add(x_reg_fl, x_reg_fl, x_reg_fl);
  __syncthreads();

  // store from register to shared
  store(o_s, x_reg_fl);
  __syncthreads();

  // store from shared to HBM
  store(g.o, o_s, {0, 0, 0, 0});
  __syncthreads();
}

void dispatch_micro(float *d_x, float *d_o, int N) {
  using _gl = gl<float, -1, -1, -1, -1, st_fl<_row, _col>>;
  using globals = micro_globals;
  _gl x_arg{d_x, 1, 1, _row, _col};
  _gl o_arg{d_o, 1, 1, _row, _col};
  globals g{x_arg, o_arg};
  unsigned long mem_size = 50480;
  hipFuncSetAttribute(reinterpret_cast<const void*>(micro_tk), hipFuncAttributeMaxDynamicSharedMemorySize,
                       mem_size);

  micro_tk<<<1, 32, mem_size>>>(g);

  hipError_t err;
  err = hipGetLastError();
  if (err != hipSuccess) {
    printf("CUDA kernel launch failed: %s\n", hipGetErrorString(err));
  }
  hipDeviceSynchronize();
}

// input_t is std::vector<float>
output_t custom_kernel(input_t data) {
  output_t result;
  hipError_t err;

  int N = data.size();
  result.resize(N);

  if (N == 0) {
    std::cout << "SIZE: " << data.size() << std::endl;
    return result;
  }

  // Allocate device memory
  float *d_input, *d_output;
  err = hipMalloc(&d_input, N * sizeof(float));
  if (err != hipSuccess) {
    printf("CUDA malloc failed for d_input: %s\n", hipGetErrorString(err));
    return result;
  }
  err = hipMalloc(&d_output, N * sizeof(float));
  if (err != hipSuccess) {
    printf("CUDA malloc failed for d_output: %s\n", hipGetErrorString(err));
    return result;
  }

  // Copy input to device
  err = hipMemcpy(d_input, data.data(), N * sizeof(float),
                   hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    printf("CUDA memcpy HostToDevice failed: %s\n", hipGetErrorString(err));
    return result;
  }

  hipDeviceSynchronize();
  CudaCheckError();
  dispatch_micro(d_input, d_output, N);
  hipDeviceSynchronize();
  CudaCheckError();

  // Copy result back to host
  err = hipMemcpy(result.data(), d_output, N * sizeof(float),
                   hipMemcpyDeviceToHost);

  if (err != hipSuccess) {
    printf("CUDA memcpy DeviceToHost failed: %s\n", hipGetErrorString(err));
    return result;
  }

  // Free device memory
  hipFree(d_input);
  hipFree(d_output);

  return result;
}
