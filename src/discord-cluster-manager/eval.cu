#include <chrono>
#include <iostream>

#include "reference.cuh"
#include "train.cuh"

#define WARMUP_RUNS 10
#define TIMED_RUNS 100

// checks that a CUDA API call returned successfully, otherwise prints an error message and exits.
static void cuda_check(hipError_t status, const char* expr, const char* file, int line, const char* function)
{
    if(status != hipSuccess) {
        std::cerr << "CUDA error (" << (int)status << ") while evaluating expression "
                  << expr << " at "
                  << file << '('
                  << line << ") in `"
                  << function << "`: "
                  << hipGetErrorString(status) << std::endl;
        // following pytest convention, exit code 3 means internal error
        std::exit(3);
    }
}

#define cuda_check(expr) cuda_check(expr, #expr, __FILE__, __LINE__, __FUNCTION__)

float measure_runtime() {
    std::cout << "warming up..." << std::endl;

    for (int i = 0; i < WARMUP_RUNS; i++) {
        auto data = generate_input();
        custom_kernel(data);
    }
    cuda_check(hipDeviceSynchronize());

    using double_duration = std::chrono::duration<double>;
    double total_duration = 0.0;

    for (int i = 0; i < TIMED_RUNS; i++) {
        auto data = generate_input();

        auto start = std::chrono::high_resolution_clock::now();
        auto submission_output = custom_kernel(data);
        cuda_check(hipDeviceSynchronize());
        auto end = std::chrono::high_resolution_clock::now();

        total_duration += std::chrono::duration_cast<double_duration>(end - start).count();

        auto reference_output = ref_kernel(data);
        if (!check_implementation(submission_output, reference_output)) {
            std::cout << "check_implementation failed" << std::endl;
            return 1;
        }

    }


    double average_duration = total_duration / TIMED_RUNS;
    std::cout << "submitted kernel runtime: " << average_duration << " seconds" << std::endl;
    return average_duration;
}

int main() {
    auto data = generate_input();
    auto reference_output = ref_kernel(data);
    auto submission_output = custom_kernel(data);

    if (!check_implementation(submission_output, reference_output)) {
        std::cout << "check_implementation failed" << std::endl;
        return 1;
    }

    float s = measure_runtime();
    if (s < 0) {
        return 1;
    }

    std::cout << "score: " << s << std::endl;

    return 0;
}
